
#include <hip/hip_runtime.h>
#include <iostream>
#include <utility> 
#include <vector>
#include <cstring> 
#include <thread>
#include <chrono>
using namespace std;
using namespace std::chrono; 

bool check_matrix(vector<int> H1, vector<int> H2, int n, int m)
{
    for (int i = 0; i < n; i++)
    {
        for(int j = 0; j < m; j++)
        {
            if (H1[i*m + j] != H2[i*m+j])
            {
                return false;
            }
        }
    }
    return true;
}

vector<int> NW(string s0, string s1, int ma, int mi, int g)
{

    int n = strlen(s0.c_str());
    int m = strlen(s1.c_str());

    vector<int> H(n*m);
    
    H[0] = 0;

    for (int i = 1; i<n; i++) {
        H[i*m] = i*g;
    }

    for (int j = 1; j<m; j++) {
        H[j] = j*g;
    }

    for (int i=1; i <n; i++)
    {
        for (int j=1; j<m; j++) 
        {
            int p;
            if (s0[i] == s1[j]) 
            {
                p = 1;
            }
            else 
            {
                p = -1;
            }
            int res = max(max(H[(i-1)*m + (j-1)] + p, H[(i-1)*m + (j)] + g), H[(i)*m + (j-1)] + g);

            H[i*m + j] = res;
        }
    }

    return H;

}

__global__ 
void initROW(int* H, int q, int r, int g) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < r)
    {
            int begin_j = (q)*index+index;
            int end_j = (q)*(index+1)+(index+1);

            while (begin_j!=end_j) 
            {
                H[begin_j] = begin_j*g;

                begin_j+=1;
            }
        }
        else
        {
            int begin_j = (q)*index+(r);
            int end_j = (q)*(index+1)+(r);
                            
            while (begin_j!=end_j) 
            {
                H[begin_j] = begin_j*g;
                begin_j+=1;
            }
    }
}

__global__ 
void initCOL(int* H, int q, int r, int m, int g) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < r)
    {
            int begin_i = (q)*index+index;
            int end_i = (q)*(index+1)+(index+1);

            while (begin_i!=end_i) 
            {
                H[begin_i*m] = begin_i*g;

                begin_i+=1;
            }
        }
        else
        {
            int begin_i = (q)*index+(r);
            int end_i = (q)*(index+1)+(r);
                            
            while (begin_i!=end_i) 
            {
                H[begin_i*m] = begin_i*g;
                begin_i+=1;
            }
    }
}

__global__ 
void compute(int* H, int i, int j, int q, int r, int m, char* s0, char* s1, int g) 
{
    int p;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < r)
    {
        int begin_i = i-(q)*index-index;
        int end_i = i-(q)*(index+1)-(index+1);
        int begin_j = j+(q)*index+index;
        int end_j = j+(q)*(index+1)+(index+1);

        while(begin_i!= end_i && begin_j!=end_j) 
        {
            if (s0[begin_i] == s1[begin_j]) 
            {
                p = 1;
            }
            else 
            {
                p = -1;
            }
            int res = max(max(H[(begin_i-1)*m + (begin_j-1)] + p, H[(begin_i-1)*m + (begin_j)] + g), H[(begin_i)*m + (begin_j-1)] + g);

            H[begin_i*m + begin_j] = res;

            begin_i-=1;
            begin_j+=1;
        }
    }
    else
    {
        int begin_i = i-(q)*index-(r);
        int end_i = i-(q)*(index+1)-(r);
        int begin_j = j+(q)*index+(r);
        int end_j = j+(q)*(index+1)+(r);

        while(begin_i!= end_i && begin_j!=end_j) 
        {
            if (s0[begin_i] == s1[begin_j]) 
            {
                p = 1;
            }
            else 
            {
                p = -1;
            }
            int res = max(max(H[(begin_i-1)*m + (begin_j-1)] + p, H[(begin_i-1)*m + (begin_j)] + g), H[(begin_i)*m + (begin_j-1)] + g);

            H[begin_i*m + begin_j] = res;

            begin_i-=1;
            begin_j+=1;
        }
    }
}


vector<int> NW_Parallel_GPU(string s0, string s1, int ma, int mi, int g)
{

    const size_t BLOCKS_NUM = 48;
    const size_t THREADS_PER_BLOCK = 256;
    const size_t num_threads = BLOCKS_NUM * THREADS_PER_BLOCK;

    int n = s0.length();
    int m = s1.length();

    int q = m/num_threads;
    int r = m%num_threads;

    int* H;
    hipMalloc(&H, n * m * sizeof(int));

    char* d_s0;
    char* d_s1;

    hipMalloc(&d_s0, n * sizeof(char));
    hipMalloc(&d_s1, m * sizeof(char));

    hipMemcpy(d_s0, s0.c_str(), n * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_s1, s1.c_str(), m * sizeof(char), hipMemcpyHostToDevice);

    initROW<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(H,q,r,g);

    hipDeviceSynchronize();

    initCOL<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(H,q,r,m,g);

    hipDeviceSynchronize();

    int i = 1;
    int j = 1;
    int num_elem;

    while(i!= n && j!= m)
    {   

        num_elem = min(i,m-j);

        q = num_elem/num_threads;
        r = num_elem%num_threads;

        compute<<<BLOCKS_NUM, THREADS_PER_BLOCK>>>(H,i,j,q,r,m,d_s0,d_s1,g);
        hipDeviceSynchronize();

        if (i!= n-1) 
        {
            i+=1;
        }
        else 
        {
            if (j!= m-1)
            {
                j+=1;
            }
            else 
            {
                i+=1;
                j+=1;
            }
        }
    }

    int* res = (int*)malloc(sizeof(int) * n * m);
    hipMemcpy(res, H, n * m * sizeof(int), hipMemcpyDeviceToHost);

    std::vector<int> result(res, res + n * m);
    
    free(res);

    hipFree(H);

    return result;
}

void printMatrix(const std::vector<int>& data, int n, int m) {

    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < m; ++j) {
            std::cout << data[i * m + j] << " ";
        }
        std::cout << "\n";
    }
}

int main() {
    std::srand(std::time(0));

    unsigned int N = 1 << 13;

    const char nucleotides[] = {'A', 'T', 'G', 'C'};

    // string s0 = "*TAGC";
    string s0 = "*";
    for (int i = 0; i < N; ++i) {
        s0 += nucleotides[std::rand() % 4];
    }

    // string s1 = "*TAGTC";
    string s1 = "*";
    for (int i = 0; i < N; ++i) {
        s1 += nucleotides[std::rand() % 4];
    }

    std::cout << "RUNNING SEQUENTIAL CODE" << std::endl;
    auto start = high_resolution_clock::now();
    vector<int> H1 = NW(s0, s1, 1, -1, -2);
    auto end = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(end - start);
    cout << "Sequential execution time: " << duration.count() << " µs" << endl;
    std::cout << "Sequential score: " << H1.back() << std::endl;

    std::cout << "\nRUNNING GPU PARALLEL CODE" << std::endl;

    start = high_resolution_clock::now();
    std::vector<int> H2 = NW_Parallel_GPU(s0, s1, 1, -1, -2);
    end = high_resolution_clock::now();
    duration = duration_cast<microseconds>(end - start);
    std::cout << "GPU execution time: " << duration.count() << " µs" << std::endl;
    std::cout << "GPU score: " << H2.back() << std::endl;

    // printMatrix(H1,strlen(s0.c_str()),strlen(s1.c_str()));
    // printf("\n");
    // printMatrix(H2,strlen(s0.c_str()),strlen(s1.c_str()));

    int n = s0.length();
    int m = s1.length();

    if (check_matrix(H1,H2,n,m))
    {
        std::cout << "\nSame Matrices" << std::endl;
    }
    else 
    {
        std::cout << "\nDifferent Matrices" << std::endl;
    }

    return 0;
}